#include "hip/hip_runtime.h"
/**
 * @file border_detection.c
 * @author Ricardo Andrés Calvo Méndez (rcalvom@unal.edu.co)
 * @author Jorge Aurelio Morales Manrique (jomorales@unal.edu.co)
 * @brief Process a picture with a border detection filter.
 * @version 1.0
 * @date 2022-05-07
 * 
 * @copyright Copyright (c) 2022
 * 
 */

// Standard input/output library
#include <stdio.h>

// Cuda Library
#include <hip/hip_runtime.h>

// Stb image library
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image/stb_image_write.h"

// Border detection library
extern "C" {
    #include "border_detection.h"
}


/**
 * @brief Grayscale Kernel. Converts a Image to a Grayscale Image using the GPU
 * 
 * @param input_image_device direction to input image on device
 * @param grayscale_image_device direction to grayscale image on device
 * @param image_size size of the grayscale image
 * @param threads_count number of threads executed
 */
__global__ void grayscale_kernel(unsigned char* input_image_device, unsigned char* grayscale_image_device, size_t image_size, int threads_count) {
    int index = (blockDim.x * blockIdx.x) + threadIdx.x; 
    int start = image_size / threads_count * index;
    int end = image_size / threads_count * (index + 1);

    for(int i = start; i < end; i++){
        *(grayscale_image_device + i) = 0.299 * *(input_image_device + 3 * i) + 0.587 * *(input_image_device + 3 * i + 1) + 0.114 * *(input_image_device + 3 * i + 2);
    }

}

/**
 * @brief Border detection Kernel. Converts a Grayscale Image to a Border detection Image using the GPU
 * 
 * @param grayscale_image_device direction to grayscale image on device
 * @param output_image_device direction to output image on device
 * @param image_size size of the grayscale image
 * @param threads_count number of threads executed
 */
__global__ void border_detection_kernel(unsigned char* grayscale_image_device, unsigned char* output_image_device, int width, int height, double filter_intensity, size_t image_size, int threads_count) {
    int index = (blockDim.x * blockIdx.x) + threadIdx.x; 
    int start = image_size / threads_count * index;
    int end = image_size / threads_count * (index + 1);

    for(int i = start; i < end; i++){
        int p11 = (i % width == 0 || i < width) ? 0 : *(grayscale_image_device + i - width - 1);
        int p12 = (i < width) ? 0 : *(grayscale_image_device + i - width);
        int p13 = (i % width == width - 1 || i < width) ? 0 : *(grayscale_image_device + i - width + 1);
        int p21 = (i % width == 0) ? 0 : *(grayscale_image_device + i - 1);
        int p22 = *(grayscale_image_device + i);
        int p23 = (i % width == width - 1) ? 0 : *(grayscale_image_device + i + 1);
        int p31 = (i % width == 0 || i + width >= width * height) ? 0 : *(grayscale_image_device + i + width - 1);
        int p32 = (i + width >= width * height) ? 0 : *(grayscale_image_device + i + width);
        int p33 = (i % width == width - 1 || i + width >= width * height) ? 0 : *(grayscale_image_device + i + width + 1);
        int result = (-filter_intensity * p11) + (-filter_intensity * p12) + (-filter_intensity * p13) + (-filter_intensity * p21) + (8 * filter_intensity * p22) + (-filter_intensity * p23) + (-filter_intensity * p31) + (-filter_intensity * p32) + (-filter_intensity * p33);
        if(result < 0){
            *(output_image_device + i) = 0;
        }else if(result > 255){
            *(output_image_device + i) = 255;
        }else{
            *(output_image_device + i) = result;
        }
    }
}

/**
 * @brief Apply the border detection filter to a given input file and write it in a given output file
 * 
 * @param input_filename Input file path
 * @param output_filename Output file path
 * @param filter_intensity Border detection filter intensity
 * @param threads_count Numer of threads to process the image
 */
void border_detection_filter(char* input_filename, char* output_filename, double filter_intensity, int threads_count){
    /**
     * @brief Load the image form file to memory
     * 
     */
    int width, height, channels;
    unsigned char* input_image = stbi_load(input_filename, &width, &height, &channels, 3);
    if(input_image == NULL){
        perror("The image couldn't be readed. Aborting");
        exit(EXIT_FAILURE);
    }

    /**
     * @brief Allocate host memory for generated images
     * 
     */
    size_t image_size = width * height;
    unsigned char* output_image = (unsigned char*) malloc(image_size);
    if(output_image == NULL){
        perror("The memory couldn't be allocated. Aborting");
        exit(EXIT_FAILURE);
    }

    /**
     * @brief Allocate device memory for the input image and generated images
     * 
     */
    unsigned char* input_image_device;
    unsigned char* grayscale_image_device;
    unsigned char* output_image_device;
    if(hipMalloc((void **) &input_image_device, image_size * 3) != hipSuccess){
        perror("The device memory couldn't be allocated. Aborting");
        exit(EXIT_FAILURE);
    }
    if(hipMalloc((void **) &grayscale_image_device, image_size) != hipSuccess){
        perror("The device memory couldn't be allocated. Aborting");
        exit(EXIT_FAILURE);
    }
    if(hipMalloc((void **) &output_image_device, image_size) != hipSuccess){
        perror("The device memory couldn't be allocated. Aborting");
        exit(EXIT_FAILURE);
    }

    /**
     * @brief Copy image from host to device
     * 
     */
    hipMemcpy(input_image_device, input_image, image_size * 3, hipMemcpyHostToDevice);

    /**
     * @brief Execute grayscale kernel and check for errors
     * 
     */
    grayscale_kernel<<<1, threads_count>>>(input_image_device, grayscale_image_device, image_size, threads_count);
    if (hipGetLastError() != hipSuccess){
        perror("Has been ocurr an error in kernel execution. Aborting");
        exit(EXIT_FAILURE);
    }

    /**
     * @brief Execute border detection kernel and check for errors
     * 
     */
    border_detection_kernel<<<1, threads_count>>>(grayscale_image_device, output_image_device, width, height, filter_intensity, image_size, threads_count);
    if (hipGetLastError() != hipSuccess){
        perror("Has been ocurr an error in kernel execution. Aborting");
        exit(EXIT_FAILURE);
    }

    /**
     * @brief Copy generated image from device to host
     * 
     */
    hipMemcpy(output_image, output_image_device, image_size, hipMemcpyDeviceToHost);

    /**
     * @brief Save the generated image to a file
     * 
     */
    int status = stbi_write_jpg(output_filename, width, height, 1, output_image, 100);
    if(status == 0){
        perror("The output image couldn't be saved. Aborting");
        exit(EXIT_FAILURE);
    }
    /**
     * @brief Free the memory used in the generation process
     * 
     */
    stbi_image_free(input_image);
    free(output_image);
    hipFree(input_image_device);
    hipFree(grayscale_image_device);
    hipFree(output_image_device);
}

#include "hip/hip_runtime.h"
/**
 * @file border_detection.c
 * @author Ricardo Andrés Calvo Méndez (rcalvom@unal.edu.co)
 * @author Jorge Aurelio Morales Manrique (jomorales@unal.edu.co)
 * @brief Process a picture with a border detection filter.
 * @version 1.0
 * @date 2022-05-07
 * 
 * @copyright Copyright (c) 2022
 * 
 */

// Standard input/output library
#include <stdio.h>

// Time library
#include <time.h>

// System time library
#include <sys/time.h>

// System wait library
#include <sys/wait.h>

// Cuda Library
#include <hip/hip_runtime.h>

// Stb image library
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image/stb_image_write.h"

// Border detection library
extern "C" {
    #include "border_detection.h"
}

// Cuda Border detection library 
extern "C" {
    #include "border_detection.cuh"
}


/**
 * @brief Grayscale Kernel. Converts a Image to a Grayscale Image using the GPU
 * 
 * @param input_image_device direction to input image on device
 * @param grayscale_image_device direction to grayscale image on device
 * @param image_size size of the grayscale image
 * @param block_count number of blocks executed
 * @param thread_count number of threads executed
 */
__global__ void grayscale_kernel(unsigned char* input_image_device, unsigned char* grayscale_image_device, size_t image_size, int block_count, int thread_count) {
    long index = (blockDim.x * blockIdx.x) + threadIdx.x; 
    long start = image_size / (block_count * thread_count) * index;
    long end = image_size / (block_count * thread_count) * (index + 1);

    for(long i = start; i < end; i++){
        *(grayscale_image_device + i) = 0.299 * *(input_image_device + 3 * i) + 0.587 * *(input_image_device + 3 * i + 1) + 0.114 * *(input_image_device + 3 * i + 2);
    }

}

/**
 * @brief Border detection Kernel. Converts a Grayscale Image to a Border detection Image using the GPU
 * 
 * @param grayscale_image_device direction to grayscale image on device
 * @param output_image_device direction to output image on device
 * @param image_size size of the grayscale image
 * @param block_count number of blocks executed
 * @param thread_count number of threads executed
 */
__global__ void border_detection_kernel(unsigned char* grayscale_image_device, unsigned char* output_image_device, int width, int height, double filter_intensity, size_t image_size, int block_count, int thread_count) {
    long index = (blockDim.x * blockIdx.x) + threadIdx.x;
    long start = image_size / (block_count * thread_count) * index;
    long end = image_size / (block_count * thread_count) * (index + 1);

    for(long i = start; i < end; i++){
        int p11 = (i % width == 0 || i < width) ? 0 : *(grayscale_image_device + i - width - 1);
        int p12 = (i < width) ? 0 : *(grayscale_image_device + i - width);
        int p13 = (i % width == width - 1 || i < width) ? 0 : *(grayscale_image_device + i - width + 1);
        int p21 = (i % width == 0) ? 0 : *(grayscale_image_device + i - 1);
        int p22 = *(grayscale_image_device + i);
        int p23 = (i % width == width - 1) ? 0 : *(grayscale_image_device + i + 1);
        int p31 = (i % width == 0 || i + width >= width * height) ? 0 : *(grayscale_image_device + i + width - 1);
        int p32 = (i + width >= width * height) ? 0 : *(grayscale_image_device + i + width);
        int p33 = (i % width == width - 1 || i + width >= width * height) ? 0 : *(grayscale_image_device + i + width + 1);
        int result = (-filter_intensity * p11) + (-filter_intensity * p12) + (-filter_intensity * p13) + (-filter_intensity * p21) + (8 * filter_intensity * p22) + (-filter_intensity * p23) + (-filter_intensity * p31) + (-filter_intensity * p32) + (-filter_intensity * p33);
        if(result < 0){
            *(output_image_device + i) = 0;
        }else if(result > 255){
            *(output_image_device + i) = 255;
        }else{
            *(output_image_device + i) = result;
        }
    }
}

/**
 * @brief Apply the border detection filter to a given input file and write it in a given output file
 * 
 * @param input_filename Input file path
 * @param output_filename Output file path
 * @param filter_intensity Border detection filter intensity
 * @param block_count Number of blocks to process the image
 * @param threads_count Number of threads to process the image
 */
void border_detection_filter(char* input_filename, char* output_filename, double filter_intensity, int block_count, int thread_count){
    /**
     * @brief Load the image form file to memory
     * 
     */
    int width, height, channels;
    unsigned char* input_image = stbi_load(input_filename, &width, &height, &channels, 3);
    if(input_image == NULL){
        perror("The image couldn't be readed. Aborting");
        exit(EXIT_FAILURE);
    }

    /**
     * @brief Allocate host memory for generated images
     * 
     */
    size_t image_size = width * height;
    unsigned char* output_image = (unsigned char*) malloc(image_size);
    if(output_image == NULL){
        perror("The memory couldn't be allocated. Aborting");
        exit(EXIT_FAILURE);
    }

    /**
     * @brief Allocate device memory for the input image and generated images
     * 
     */
    unsigned char* input_image_device;
    unsigned char* grayscale_image_device;
    unsigned char* output_image_device;
    if(hipMalloc((void **) &input_image_device, image_size * 3) != hipSuccess){
        perror("The device memory couldn't be allocated. Aborting");
        exit(EXIT_FAILURE);
    }
    if(hipMalloc((void **) &grayscale_image_device, image_size) != hipSuccess){
        perror("The device memory couldn't be allocated. Aborting");
        exit(EXIT_FAILURE);
    }
    if(hipMalloc((void **) &output_image_device, image_size) != hipSuccess){
        perror("The device memory couldn't be allocated. Aborting");
        exit(EXIT_FAILURE);
    }

    /**
     * @brief Copy image from host to device
     * 
     */
    hipMemcpy(input_image_device, input_image, image_size * 3, hipMemcpyHostToDevice);

    /**
     * @brief Execute grayscale kernel and check for errors
     * 
     */
    struct timeval start, end;
    double stopwatch;
    gettimeofday(&start, NULL);

    grayscale_kernel<<<block_count, thread_count>>>(input_image_device, grayscale_image_device, image_size, block_count, thread_count);
    hipDeviceSynchronize();
    if (hipGetLastError() != hipSuccess){
        perror("Has been ocurr an error in kernel execution. Aborting");
        exit(EXIT_FAILURE);
    }

    /**
     * @brief Execute border detection kernel and check for errors
     * 
     */
    border_detection_kernel<<<block_count, thread_count>>>(grayscale_image_device, output_image_device, width, height, filter_intensity, image_size, block_count, thread_count);
    hipDeviceSynchronize();
    if (hipGetLastError() != hipSuccess){
        perror("Has been ocurr an error in kernel execution. Aborting");
        exit(EXIT_FAILURE);
    }

    gettimeofday(&end, NULL);
    stopwatch = (double)(end.tv_sec + (double) end.tv_usec / 1000000) - (double)(start.tv_sec + (double) start.tv_usec / 1000000);

    printf("The program has finished sucessfully.\n");
    printf("Time execution: %f\n", stopwatch);

    /**
     * @brief Copy generated image from device to host
     * 
     */
    hipMemcpy(output_image, output_image_device, image_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    /**
     * @brief Save the generated image to a file
     * 
     */
    int status = stbi_write_jpg(output_filename, width, height, 1, output_image, 100);
    if(status == 0){
        perror("The output image couldn't be saved. Aborting");
        exit(EXIT_FAILURE);
    }
    /**
     * @brief Free the memory used in the generation process
     * 
     */
    stbi_image_free(input_image);
    free(output_image);
    hipFree(input_image_device);
    hipFree(grayscale_image_device);
    hipFree(output_image_device);
}

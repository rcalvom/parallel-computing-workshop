#include "hip/hip_runtime.h"
/**
 * @file matrix_multiplication.c
 * @author Ricardo Andrés Calvo Méndez (rcalvom@unal.edu.co)
 * @author Jorge Aurelio Morales Manrique (jomorales@unal.edu.co)
 * @brief Perform matrix multiplication.
 * @version 1.0
 * @date 2022-06-06
 * 
 * @copyright Copyright (c) 2022
 * 
 */

// Standard library
#include <stdlib.h>

// Standard input/output library
#include <stdio.h>

// Time library
#include <time.h>

// System time library
#include <sys/time.h>

// System wait library
#include <sys/wait.h>

// Cuda Library
#include <hip/hip_runtime.h>

// Matrix multiplication library
extern "C" {
    #include "matrix_multiplication.h"
}

// Cuda Matrix Multiplication library 
extern "C" {
    #include "matrix_multiplication.cuh"
}


/**
 * @brief Multiply Matrices Cuda Kernel
 * 
 * @param matrix_a Matrix A
 * @param matrix_b Matrix B
 * @param matrix_c Matrix C
 * @param matrix_size Matrix size
 * @param block_count Number of Threds executed
 */
__global__ void multiply_matrices_kernel(int* matrix_a, int* matrix_b, int* matrix_c, int matrix_size, const int block_count){

    int block_col = blockIdx.x;
    int block_row = blockIdx.y;

    int* sumbatrix_c = matrix_c + matrix_size * block_count * block_row + block_count * block_col;

    int value = 0;

    int thread_row = threadIdx.y;
    int thread_col = threadIdx.x;

    for (int i = 0; i < (matrix_size / block_count); i++) {
        int* sumbatrix_a = matrix_a + matrix_size * block_count * block_row + block_count * i;
        int* sumbatrix_b = matrix_b + matrix_size * block_count * i + block_count * block_col;

        __shared__ int shared_a[40][40];
        __shared__ int shared_b[40][40];

        shared_a[thread_row][thread_col] = *(sumbatrix_a + thread_row * matrix_size + thread_col);
        shared_b[thread_row][thread_col] = *(sumbatrix_b + thread_row * matrix_size + thread_col);

        __syncthreads();

        for (int j = 0; j < block_count; j++){
            value += shared_a[thread_row][j] * shared_b[j][thread_col];
        }

        __syncthreads();

    }
    *(sumbatrix_c + thread_row * matrix_size + thread_col) = value;
}


/**
 * @brief Perform Matrix multiplication
 * 
 * @param matrix_a_filename First matrix for multiplication
 * @param matrix_b_filename Second matrix for multiplication
 * @param matrix_b_filename Produced matrix by multiplication
 * @param matrix_size Matrix size
 * @param block_count Number of Threds executed
 */
void matrix_multiplication(char * matrix_a_filename, char* matrix_b_filename, char* matrix_c_filename, int matrix_size, int block_count){
    /**
     * @brief Generate matrices with random data
     * 
     */
    generate_matrix(matrix_a_filename, matrix_size);
    generate_matrix(matrix_b_filename, matrix_size);

    /**
     * @brief Load matrices from files
     * 
     */
    int* matrix_a = read_matrix(matrix_a_filename, matrix_size);
    int* matrix_b = read_matrix(matrix_b_filename, matrix_size);

    /**
     * @brief Allocate memory for matrices in device
     * 
     */

    int* matrix_a_device;
    int* matrix_b_device;
    int* matrix_c_device;
    if(hipMalloc((void **) &matrix_a_device, sizeof(int) * matrix_size * matrix_size) != hipSuccess){
        perror("The device memory couldn't be allocated. Aborting");
        exit(EXIT_FAILURE);
    }
    if(hipMalloc((void **) &matrix_b_device, sizeof(int) * matrix_size * matrix_size) != hipSuccess){
        perror("The device memory couldn't be allocated. Aborting");
        exit(EXIT_FAILURE);
    }
    if(hipMalloc((void **) &matrix_c_device, sizeof(int) * matrix_size * matrix_size) != hipSuccess){
        perror("The device memory couldn't be allocated. Aborting");
        exit(EXIT_FAILURE);
    }

    /**
     * @brief Copy matrices to device, Multiply matrices and copy produced matrices to host
     * 
     */

    struct timeval start, end;
    double stopwatch;
    gettimeofday(&start, NULL);

    hipMemcpy(matrix_a_device, matrix_a, sizeof(int) * matrix_size * matrix_size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    hipMemcpy(matrix_b_device, matrix_b, sizeof(int) * matrix_size * matrix_size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    dim3 dimBlock(block_count, block_count);
    dim3 dimGrid(matrix_size / dimBlock.x, matrix_size / dimBlock.y);

    multiply_matrices_kernel<<<dimGrid, dimBlock>>>(matrix_a_device, matrix_b_device, matrix_c_device, matrix_size, block_count);
    hipDeviceSynchronize();
    if (hipGetLastError() != hipSuccess){
        perror("Has been ocurr an error in kernel execution. Aborting");
        exit(EXIT_FAILURE);
    }

    int* matrix_c = (int*) malloc(sizeof(int) * matrix_size * matrix_size);
    hipMemcpy(matrix_c, matrix_c_device, sizeof(int) * matrix_size * matrix_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    gettimeofday(&end, NULL);
    stopwatch = (double)(end.tv_sec + (double) end.tv_usec / 1000000) - (double)(start.tv_sec + (double) start.tv_usec / 1000000);

    printf("The matrix multiplication has finished.\n");
    printf("Time execution: %f\n", stopwatch);

    /**
     * @brief Save result
     * 
     */
    save_matrix(matrix_c, matrix_c_filename, matrix_size);

    /**
     * @brief Free Allocated Memory
     * 
     */
    free(matrix_a);
    free(matrix_b);
    free(matrix_c);
    hipFree(matrix_a_device);
    hipFree(matrix_b_device);
    hipFree(matrix_c_device);
}

/**
 * @brief Generate matrix with random numbers and saves it to a TXT file.
 * Columns are separated by spaces
 * Rows are separated by new lines
 * 
 * @param matrix_size Matrix size
 * @return Generated matrix
 */
void generate_matrix(char* matrix_filename, int matrix_size) {
    FILE* file = fopen(matrix_filename, "w");
    if(file == NULL){
        perror("The file couldn't be opened. Aborting");
        exit(EXIT_FAILURE);
    }
    for (int i = 0; i < matrix_size; i++) {
        for (int j = 0; j < matrix_size; j++) {
            int element = (rand() % 40);
            fprintf(file, "%d ", element);
        }
        fprintf(file, "\n");
    }
    fclose(file);
}

/**
 * @brief Load Matrix from TXT file.
 * Columns are separated by spaces
 * Rows are separated by new lines
 * 
 * @param matrix_filename Matrix filename
 * @param matrix_size Matrix size
 * @return Matrix Readed
 */
int* read_matrix(char* matrix_filename, int matrix_size){
    int* matrix = (int*) malloc(sizeof(int) * matrix_size * matrix_size);
    if(matrix == NULL){
        perror("The memory couldn't be allocated. Aborting");
        exit(EXIT_FAILURE);
    }
    FILE* file = fopen(matrix_filename, "r");
    if(file == NULL){
        perror("The file couldn't be opened. Aborting");
        exit(EXIT_FAILURE);
    }
    for(int i = 0; i < matrix_size * matrix_size; i++){
        fscanf(file, "%i", matrix + i);
    }
    fclose(file);
    return matrix;
}

/**
 * @brief Save a given matrix to a file
 * 
 * @param matrix Matrix to save
 * @param matrix_filename File to write
 * @param matrix_size Matrix size
 */
void save_matrix(int* matrix, char* matrix_filename, int matrix_size){
    FILE* file = fopen(matrix_filename, "w+");
    if(file == NULL){
        perror("The file couldn't be created. Aborting");
        exit(EXIT_FAILURE);
    }
    for(int i = 0; i < matrix_size; i++){
        for(int j = 0; j < matrix_size; j++){
            fprintf(file, "%i ", *(matrix + (i * matrix_size) + j));
        }
        fprintf(file, "\n");
    }
    fclose(file);
}
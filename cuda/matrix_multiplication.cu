#include "hip/hip_runtime.h"
/**
 * @file matrix_multiplication.c
 * @author Ricardo Andrés Calvo Méndez (rcalvom@unal.edu.co)
 * @author Jorge Aurelio Morales Manrique (jomorales@unal.edu.co)
 * @brief Perform matrix multiplication.
 * @version 1.0
 * @date 2022-06-06
 * 
 * @copyright Copyright (c) 2022
 * 
 */

// Standard library
#include <stdlib.h>

// Standard input/output library
#include <stdio.h>

// Time library
#include <time.h>

// System time library
#include <sys/time.h>

// System wait library
#include <sys/wait.h>

// Cuda Library
#include <hip/hip_runtime.h>

// Matrix multiplication library
extern "C" {
    #include "matrix_multiplication.h"
}

// Cuda Matrix Multiplication library 
extern "C" {
    #include "matrix_multiplication.cuh"
}


__global__ void multiply_matrices_kernel(int* matrix_a, int* matrix_b, int* matrix_c, int matrix_size, int block_count, int thread_count){

}


/**
 * @brief Perform Matrix multiplication
 * 
 * @param matrix_a_filename First matrix for multiplication
 * @param matrix_b_filename Second matrix for multiplication
 * @param matrix_b_filename Produced matrix by multiplication
 * @param matrix_size Matrix size
 * @param block_count Number of Threds executed
 * @param thread_count Number of Threds executed
 */
void matrix_multiplication(char * matrix_a_filename, char* matrix_b_filename, char* matrix_c_filename, int matrix_size, int block_count, int thread_count){

    /**
     * @brief Load matrices from files
     * 
     */
    int* matrix_a = read_matrix(matrix_a_filename, matrix_size);
    int* matrix_b = read_matrix(matrix_b_filename, matrix_size);

    /**
     * @brief Allocate memory for matrices in device
     * 
     */

    int* matrix_a_device;
    int* matrix_b_device;
    int* matrix_c_device;
    if(hipMalloc((void **) &matrix_a_device, sizeof(int) * matrix_size * matrix_size) != hipSuccess){
        perror("The device memory couldn't be allocated. Aborting");
        exit(EXIT_FAILURE);
    }
    if(hipMalloc((void **) &matrix_b_device, sizeof(int) * matrix_size * matrix_size) != hipSuccess){
        perror("The device memory couldn't be allocated. Aborting");
        exit(EXIT_FAILURE);
    }
    if(hipMalloc((void **) &matrix_c_device, sizeof(int) * matrix_size * matrix_size) != hipSuccess){
        perror("The device memory couldn't be allocated. Aborting");
        exit(EXIT_FAILURE);
    }

    /**
     * @brief Copy matrices to device, Multiply matrices and copy produced matrices to host
     * 
     */

    struct timeval start, end;
    double stopwatch;
    gettimeofday(&start, NULL);

    hipMemcpy(matrix_a_device, matrix_a, sizeof(int) * matrix_size * matrix_size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    hipMemcpy(matrix_b_device, matrix_b, sizeof(int) * matrix_size * matrix_size, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    multiply_matrices_kernel<<<block_count, thread_count>>>(matrix_a_device, matrix_b_device, matrix_c_device, matrix_size, block_count, thread_count);
    hipDeviceSynchronize();
    if (hipGetLastError() != hipSuccess){
        perror("Has been ocurr an error in kernel execution. Aborting");
        exit(EXIT_FAILURE);
    }

    hipMemcpy(matrix_c, matrix_c_device, sizeof(int) * matrix_size * matrix_size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    gettimeofday(&end, NULL);
    stopwatch = (double)(end.tv_sec + (double) end.tv_usec / 1000000) - (double)(start.tv_sec + (double) start.tv_usec / 1000000);

    printf("The matrix multiplication has finished.\n");
    printf("Time execution: %f\n", stopwatch);

    /**
     * @brief Save result
     * 
     */
    save_matrix(matrix_c, matrix_c_filename, matrix_size);

    /**
     * @brief Free Allocated Memory
     * 
     */
    free(matrix_a);
    free(matrix_b);
    free(matrix_c);
    hipFree(matrix_a_device);
    hipFree(matrix_b_device);
    hipFree(matrix_c_device);
}

/**
 * @brief Generate matrix with random numbers and saves it to a TXT file.
 * Columns are separated by spaces
 * Rows are separated by new lines
 * 
 * @param matrix_size Matrix size
 * @return Generated matrix
 */
void generate_matrix(char* matrix_filename, int matrix_size) {
    FILE* file = fopen(matrix_filename, "w");
    if(file == NULL){
        perror("The file couldn't be opened. Aborting");
        exit(EXIT_FAILURE);
    }
    for (int i = 0; i < matrix_size; i++) {
        for (int j = 0; j < matrix_size; j++) {
            int element = (rand() % 40);
            fprintf(file, "%d ", element);
        }
        fprintf(file, "\n");
    }
    fclose(file);
}

/**
 * @brief Load Matrix from TXT file.
 * Columns are separated by spaces
 * Rows are separated by new lines
 * 
 * @param matrix_filename Matrix filename
 * @param matrix_size Matrix size
 * @return Matrix Readed
 */
int* read_matrix(char* matrix_filename, int matrix_size){
    int* matrix = malloc(sizeof(int) * matrix_size * matrix_size);
    if(matrix == NULL){
        perror("The memory couldn't be allocated. Aborting");
        exit(EXIT_FAILURE);
    }
    FILE* file = fopen(matrix_filename, "r");
    if(file == NULL){
        perror("The file couldn't be opened. Aborting");
        exit(EXIT_FAILURE);
    }
    for(int i = 0; i < matrix_size * matrix_size; i++){
        fscanf(file, "%i", matrix + i);
    }
    fclose(file);
    return matrix;
}

/**
 * @brief Multiply two matrices A and B
 * 
 * @param matrix_a Matrix A
 * @param matrix_b Matrix B
 * @param matrix_size Matrix size
 * @param thread_count Number of Threads to excecute
 * @return Matrix C (A * B)
 */
int* multiply_matrices(int* matrix_a, int* matrix_b, int matrix_size, int thread_count){
    int* matrix_c = malloc(sizeof(int) * matrix_size * matrix_size);
    if(matrix_c == NULL){
        perror("The memory couldn't be allocated. Aborting");
        exit(EXIT_FAILURE);
    }
    omp_set_num_threads(thread_count);
    #pragma omp parallel for
    for(int index = 0; index < matrix_size * matrix_size; index++){
        int i = index / matrix_size;
        int j = index % matrix_size;
        int value = 0;
        for(int k = 0; k < matrix_size; k++){
            value += *(matrix_a + ( i * matrix_size) + k) * *(matrix_b + (matrix_size * k) + j);
        }
        *(matrix_c + (i * matrix_size) + j) = value;
    }
    return matrix_c;
}

/**
 * @brief Save a given matrix to a file
 * 
 * @param matrix Matrix to save
 * @param matrix_filename File to write
 * @param matrix_size Matrix size
 */
void save_matrix(int* matrix, char* matrix_filename, int matrix_size){
    FILE* file = fopen(matrix_filename, "w+");
    if(file == NULL){
        perror("The file couldn't be created. Aborting");
        exit(EXIT_FAILURE);
    }
    for(int i = 0; i < matrix_size; i++){
        for(int j = 0; j < matrix_size; j++){
            fprintf(file, "%i ", *(matrix + (i * matrix_size) + j));
        }
        fprintf(file, "\n");
    }
    fclose(file);
}